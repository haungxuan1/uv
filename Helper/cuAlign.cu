#include "hip/hip_runtime.h"
#include "pch.h"
#include "Helper.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using namespace cv;
using namespace std;

namespace Align
{
	enum {ePT_Foreground = 0, ePT_Diameter, ePT_Stride, ePT_Range, ePT_Erode, ePT_Dilate};

	void PreImage(Mat& cImIn, Mat& cImBin, Mat& cImOut, int nForeground, int nErode, int nDilate)
	{
		GaussianBlur(cImIn, cImBin, cv::Size(7, 7), 1.5, 0);

		threshold(cImBin, cImBin, nForeground, 255, THRESH_BINARY_INV);

		Mat kErode = Mat::ones(nErode, nErode, CV_8UC1);

		Mat kDilate = Mat::ones(nDilate, nDilate, CV_8UC1);

		erode(cImBin, cImOut, kErode);

		dilate(cImOut, cImOut, kDilate);
	}

	vector<Rect> ImConnected(Mat& cIm)
	{
		Mat labels = Mat::zeros(cIm.rows, cIm.cols, CV_32S), stats, centroids;

		connectedComponentsWithStats(cIm, labels, stats, centroids);

		auto lpCon = (int*)stats.data;

		vector<Rect> vCon(stats.rows - 1);

		for (int y = 1; y < stats.rows; ++y)
		{
			vCon[y - 1].x = lpCon[y * 5 + CC_STAT_LEFT];

			vCon[y - 1].y = lpCon[y * 5 + CC_STAT_TOP];

			vCon[y - 1].width = lpCon[y * 5 + CC_STAT_WIDTH];

			vCon[y - 1].height = lpCon[y * 5 + CC_STAT_HEIGHT];
		}
		return vCon;
	}

	inline void eraseLargerArea(vector<Rect>& vCon, int nDiameter) // ����connectedComponentsWithStats��������� // �ò�����������²�������
	{
		for (auto itr = vCon.begin(); itr != vCon.end(); )
		{
			if (itr->height < nDiameter && itr->width < nDiameter) { ++itr; continue; }

			itr = vCon.erase(itr);
		}
	}

	inline void eraseSmallerArea(vector<Rect>& vCon, double dTh)
	{
		if (vCon.empty()) return;

		auto iThArea = max_element(vCon.begin(), vCon.end(), [](Rect& rt1, Rect& rt2) {return rt1.area() < rt2.area(); })->area() * dTh;

		for (auto itr = vCon.begin(); itr != vCon.end();)
		{
			if (iThArea < itr->area()) { ++itr; continue; }

			itr = vCon.erase(itr);
		}
	}

	inline Rect rtUnite(Rect& rt1, Rect rt2)
	{
		return (rt1 & rt2);
	}

	inline Rect rtUnion(Rect& rt1, Rect rt2)
	{
		return (rt1 | rt2);
	}

	void ImUnion(vector<Rect>& vRect) // �ò�����������²�������
	{
		bool bUnite = false;
		do
		{
			bUnite = false;

			for (auto itr = vRect.begin(); itr != vRect.end(); ++itr)
			{
				auto itr2 = itr; ++itr2;

				for (; itr2 != vRect.end();)
				{
					if (rtUnite(*itr, *itr2).empty()) { ++itr2; continue; }

					(*itr) = rtUnion(*itr, *itr2);

					itr2 = vRect.erase(itr2);

					bUnite = true;
				}
			}
		} while (bUnite);
	}

	void eraseIsolatedPoints(int nDiameter, vector<Rect>& vConnected)
	{
		eraseLargerArea(vConnected, nDiameter);// �ų��ϴ�Ĳ����

		ImUnion(vConnected);

		eraseSmallerArea(vConnected, 0.5); // �ų���С�Ĳ����
	}

	vector<tuple<Rect, Rect, int>> ImUnite(vector<Rect>& rt1, vector<Rect>& rt2, int nStride)
	{
		cout << "nStride: " << nStride << endl;

		Rect rtTmp; vector<tuple<Rect, Rect, int>> vUnite;

		for (int i = 0; i < rt1.size(); ++i)
		{
			for (int j = 0; j < rt2.size(); ++j)
			{
				rtTmp = rt2[j]; rtTmp.y += nStride;

				auto iUnite = rtUnite(rt1[i], rtTmp);

				if (iUnite.empty()) continue;

				vUnite.push_back(make_tuple(rt1[i], rt2[j], iUnite.area()));
			}
		}
		return vUnite;
	}

	bool ImUnite(Mat& cIm1, Mat& cIm2, int nDiameter, int nStride, array<Rect, 2>& vImUnite)
	{
		vector<Rect> vCon1 = ImConnected(cIm1);

		vector<Rect> vCon2 = ImConnected(cIm2);

		eraseIsolatedPoints(nDiameter, vCon1);

		eraseIsolatedPoints(nDiameter, vCon2);

		vector<tuple<Rect, Rect, int>> vUnite = ImUnite(vCon1, vCon2, nStride);

		if (vUnite.empty()) { cout << __FUNCTION__ << ": " << "empty" << endl; return false; }

		auto iMaxUnite = max_element(vUnite.begin(), vUnite.end(),
			[](tuple<Rect, Rect, int>& u1, tuple<Rect, Rect, int>& u2)
			{ return get<2>(u1) < get<2>(u2); });

		auto& iCon1 = get<0>(*iMaxUnite), & iCon2 = get<1>(*iMaxUnite);

		int w = min(iCon1.width, iCon2.width), h = min(iCon1.height, iCon2.height);

		if (!!iCon2.y)	// ��������
		{
			vImUnite[0].x = iCon1.x, vImUnite[0].y = iCon1.y;

			vImUnite[0].width = w, vImUnite[0].height = h;

			vImUnite[1].x = iCon2.x, vImUnite[1].y = iCon2.y;

			vImUnite[1].width = w, vImUnite[1].height = h;
		}
		else  //�ײ�����
		{
			vImUnite[0].x = iCon1.x, vImUnite[0].y = iCon1.y + iCon1.height - h;

			vImUnite[0].width = w, vImUnite[0].height = h;

			vImUnite[1].x = iCon2.x, vImUnite[1].y = iCon2.y + iCon2.height - h;

			vImUnite[1].width = w, vImUnite[1].height = h;
		}
		return true;
	}

	__global__ void cuCorrect(
		float* a, float* b,
		int cols, int rows,
		int nCompensation,
		float* aIntegral, float* bIntegral,
		double* score)
	{
		int ox = threadIdx.x + nCompensation;

		int oy = blockIdx.x + nCompensation;

		int x1 = max(0, ox), x2 = max(0, -ox);

		int y1 = max(0, oy), y2 = max(0, -oy);

		int w = cols - abs(ox), h = rows - abs(oy);

		double dAB = 0.0;

		for (int y = 0; y < h; ++y)
		{
			for (int x = 0; x < w; ++x)
			{
				int idx1 = (y1 + y) * cols + (x1 + x);

				int idx2 = (y2 + y) * cols + (x2 + x);

				dAB += (a[idx1] * b[idx2]);
			}
		}

		int aw = cols + 1, ah = rows + 1;

		double dA =
			aIntegral[(y1 + 1) * aw + (x1 + 1)] +
			aIntegral[(y1 + h) * aw + (x1 + w)] -
			aIntegral[(y1 + h) * aw + (x1 + 1)] -
			aIntegral[(y1 + 1) * aw + (x1 + w)];

		double dB =
			bIntegral[(y2 + h) * aw + (x2 + w)] +
			bIntegral[(y2 + 1) * aw + (x2 + 1)] -
			bIntegral[(y2 + h) * aw + (x2 + 1)] -
			bIntegral[(y2 + 1) * aw + (x2 + w)];

		score[blockIdx.x * blockDim.x + threadIdx.x] = dAB / sqrt(dA * dB);
	}

	array<int, 2> cuAlign(
		Mat& a, Mat& b,
		Mat& aIntegral, Mat& bIntegral,
		int nRange, double& dScore) // 210ms
	{
		Mat cScore(nRange * 2, nRange * 2, CV_64FC1);

		float* cuA, * cuB, * cuAcIntegral, * cuBIntegral; double* cuScore;

		hipMalloc<double>(&cuScore, sizeof(double) * cScore.rows * cScore.cols);

		hipMalloc<float>(&cuA, sizeof(float) * a.rows * a.cols);

		hipMalloc<float>(&cuB, sizeof(float) * a.rows * a.cols);

		hipMalloc<float>(&cuAcIntegral, sizeof(float) * aIntegral.rows * aIntegral.cols);

		hipMalloc<float>(&cuBIntegral, sizeof(float) * aIntegral.rows * aIntegral.cols);

		hipMemcpy(cuA, a.data, sizeof(float) * a.rows * a.cols, hipMemcpyHostToDevice);

		hipMemcpy(cuB, b.data, sizeof(float) * a.rows * a.cols, hipMemcpyHostToDevice);

		hipMemcpy(cuAcIntegral, aIntegral.data, sizeof(float) * aIntegral.rows * aIntegral.cols, hipMemcpyHostToDevice);

		hipMemcpy(cuBIntegral, bIntegral.data, sizeof(float) * aIntegral.rows * aIntegral.cols, hipMemcpyHostToDevice);

		cuCorrect << <nRange * 2, nRange * 2 >> > (
			cuA, cuB,
			a.cols, a.rows,
			-nRange,
			cuAcIntegral, cuBIntegral,
			cuScore);

		hipMemcpy(cScore.data, cuScore, sizeof(double) * cScore.rows * cScore.cols, hipMemcpyDeviceToHost);

		double* lpMax = max_element<double*>((double*)cScore.data, (double*)(cScore.data + cScore.rows * cScore.cols * sizeof(double)));

		int idxMax = lpMax - (double*)cScore.data; dScore = *lpMax;

		hipFree(cuA);

		hipFree(cuB);

		hipFree(cuAcIntegral);

		hipFree(cuBIntegral);

		hipFree(cuScore);

		return array<int, 2>{ idxMax% cScore.cols - nRange, idxMax / cScore.cols - nRange };
	}

	array<int, 2> ImAlign(Mat& a, Mat& b, int nRange, double& dScore)
	{
		a.convertTo(a, CV_32F);	b.convertTo(b, CV_32F);

		Mat a2 = a.mul(a), b2 = b.mul(b);

		Mat aIntegral, bIntegral;

		integral(a2, aIntegral, CV_32FC1);

		integral(b2, bIntegral, CV_32FC1);

		return cuAlign(a, b, aIntegral, bIntegral, nRange, dScore);
	}
}
/******************************************************************************************/
// ��ɫ���������Բֱ����ͼ������n*pass����������Χ����ʴ�ˣ�30�������ͺˣ�80��
void cuAlign(
	cv::Mat& cIm1, cv::Mat& cIm2,
	cv::Mat& cImBin1, cv::Mat& cImBin2,
	cv::Mat& cImAlign1, cv::Mat& cImAlign2,
	cv::Mat& cImUnite1, cv::Mat& cImUnite2,
	std::array<cv::Rect, 2>& rtImUnite, std::array<int, 2>& vAlign,
	double& dScore, double& dTime, 
	std::array<int, 6> vParam, std::array<int, 2>& vInterval)
{
	Align::PreImage(cIm1, cImBin1, cImAlign1, vParam[Align::ePT_Foreground], vParam[Align::ePT_Erode], vParam[Align::ePT_Dilate]);

	Align::PreImage(cIm2, cImBin2, cImAlign2, vParam[Align::ePT_Foreground], vParam[Align::ePT_Erode], vParam[Align::ePT_Dilate]);

	if (!Align::ImUnite(cImAlign1, cImAlign2, vParam[Align::ePT_Diameter], vParam[Align::ePT_Stride], rtImUnite)) { vInterval = { 0, vParam[2] }; return; }

	cImUnite1 = cImBin1(rtImUnite[0]).clone();

	cImUnite2 = cImBin2(rtImUnite[1]).clone();

	vAlign = Align::ImAlign(cImUnite1, cImUnite2, vParam[Align::ePT_Range], dScore);

	vInterval[0] = rtImUnite[0].x - rtImUnite[1].x + vAlign[0];

	vInterval[1] = rtImUnite[0].y - rtImUnite[1].y + vAlign[1];
}
