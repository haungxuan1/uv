#include "hip/hip_runtime.h"
#include "Helper.h"
#include <hip/hip_runtime.h>
#include <>

using namespace cv;
using namespace std;

typedef int ImType;

void ImEncode(Mat& cImage, Mat& cEncode)
{
	// ��x����ѹ��
	ImType idx1 = 0, idx2 = 0;

	for (ImType y = 0; y < cEncode.rows; ++y)
	{
		for (ImType x = 0; x < cEncode.cols; ++x)
		{
			idx2 = y * cEncode.cols + x;

			for (int i = 0; i < 8; ++i)
			{
				idx1 = y * cImage.cols + x * 8 + i;

				cEncode.data[idx2] |= ((cImage.data[idx1] & 0x1) << i);
			}
		}
	}
}

void ImDecode(Mat& cEncode, Mat& cDecode)
{
	ImType idx1 = 0, idx2 = 0;

	for (ImType y = 0; y < cEncode.rows; ++y)
	{
		for (ImType x = 0; x < cEncode.cols; ++x)
		{
			idx2 = y * cEncode.cols + x;

			for (int i = 0; i < 8; ++i)
			{
				idx1 = y * cDecode.cols + x * 8 + i;

				cDecode.data[idx1] = ((cEncode.data[idx2] >> i) & 0x1) * 0xff;
			}
		}
	}
}

struct tagImPoint
{
	ImType x, y, b, idx; // b = x ����encode bit ����
	__device__ tagImPoint() :x(0), y(0), idx(0){}
	__device__ tagImPoint(ImType _x, ImType _y, ImType _b) : x(_x), y(_y), b(_b), idx(0) {}
};

__global__ void cuZSSynchro(
	uchar* lpImage,
	uchar* lpSign,
	ImType sw, ImType sh/*�߳�������*/)
{
	ImType w = sw * gridDim.x + 2;

	ImType h = sh * blockDim.x + 2;

	uchar s[] = { 0, 0 };

	tagImPoint o(sw * blockIdx.x, sh * threadIdx.x, 0), p; // ��������ʼ��

	for (ImType x = 0; x < sw; ++x)
	{
		for (ImType y = 0; y < sh; ++y)
		{
			p.x = o.x + x + 1;

			p.y = o.y + y + 1; // os

			p.idx = p.y * w + p.x;

			for (ImType b = 0; b < 8; ++b)
			{
				s[0] = (lpImage[p.idx] >> b & 0x1);

				lpImage[p.idx] = lpImage[p.idx] & (~(0x1 << b));

				lpImage[p.idx] |= (s[lpSign[p.idx] >> b & 0x1] << b);
			}
		}
	}
}

__global__ void cuZSKernel1(
	uchar* lpImage,
	uchar* lpSign,
	uchar* lpRef/*ˢ�±�־*/,
	ImType sw, ImType sh/*�߳�������*/)
{
	// ��^�滻�߼��ж�
	int bit[2] = { 8, 1 };

	int begin = 0, end = 0;

	ImType w = sw * gridDim.x + 2;

	ImType h = sh * blockDim.x + 2;

	int sum = 0, changed = 0;

	uchar sync[6] = { 0 };

	tagImPoint p[9] = { tagImPoint() }; // �����

	tagImPoint o(sw * blockIdx.x, sh * threadIdx.x, (ImType)0); // ��������ʼ��

	// ���򲹳�
	tagImPoint c[9] = {
		{ 0, 0, 0 }, { 0, -1, 0 }, { 0, -1, 1 }, 
		{ 0, 0, 1 }, { 0, 1, 1 }, { 0, 1, 0 }, 
		{ 0, 1, -1 }, { 0, 0, -1 }, { 0, -1, -1 } };

	for (ImType x = 0; x < sw; ++x)
	{
		for (ImType y = 0; y < sh; ++y)
		{
			p[0].x = o.x + x + 1;

			p[0].y = o.y + y + 1;// os

			p[0].idx = p[0].y * w + p[0].x;

			for (ImType b = 1; b <= 8; ++b)
			{
				p[0].b = b;

				for (int i = 1; i < 9; ++i)
				{
					p[i].x = p[0].x + c[i].x;

					p[i].y = p[0].y + c[i].y;

					p[i].b = p[0].b + c[i].b;
				
					// b = 0 -> b = 8, x = x -1
					// b = 9 -> b = 1, x = x + 1
					begin = !!!p[i].b;

					end = !(p[i].b ^ 9);

					p[i].x -= begin;

					p[i].x += end;

					p[i].b = (begin * bit[0]) ^ p[i].b;

					bit[0] = (begin * bit[0]) ^ p[i].b;

					p[i].b = (begin * bit[0]) ^ p[i].b;

					p[i].b = (end * bit[1]) ^ p[i].b;

					bit[1] = (end * bit[1]) ^ p[i].b;

					p[i].b = (end * bit[1]) ^ p[i].b;

					p[i].idx = p[i].y * w + p[i].x;

					sum += (lpImage[p[i].idx] >> (p[i].b - 1) & 0x1);

					bit[0] = 8; bit[1] = 1;
				} // 8����

				changed += max(0, (lpImage[p[1].idx] >> (p[1].b - 1) & 0x1) -  (lpImage[p[8].idx] >> (p[8].b - 1) & 0x1));

				for (int i = 2; i < 9; ++i)
				{
					changed += max(0, (lpImage[p[i].idx] >> (p[i].b - 1) & 0x1) - (lpImage[p[i-1].idx] >> (p[i-1].b - 1) & 0x1));
				}

				sync[0] = !((lpImage[p[0].idx] >> (p[0].b - 1) & 0x1) ^1);// p1 = 1
				sync[1] = !(sum ^ min(max(2, sum), 6)); // 2 <= sum <=6
				sync[2] = !(changed ^ 1);// changed = 1
				sync[3] = !((lpImage[p[1].idx] >> (p[1].b - 1) & 0x1)*
					(lpImage[p[3].idx] >> (p[3].b - 1) & 0x1)*
					(lpImage[p[5].idx] >> (p[5].b - 1) & 0x1));// p2*p4*p6 = 0
				sync[4] = !((lpImage[p[7].idx] >> (p[7].b - 1) & 0x1) *
					(lpImage[p[3].idx] >> (p[3].b - 1) & 0x1) *
					(lpImage[p[5].idx] >> (p[5].b - 1) & 0x1));// p4*p6*p8 = 0

				sync[5] = (sync[0] & sync[1] & sync[2] & sync[3] & sync[4]);

				lpSign[p[0].idx] |= (sync[5] << (p[0].b - 1));

				lpRef[p[0].idx] |= (sync[5] << (p[0].b - 1));

				sum = 0, changed = 0;;
			}
		}
	}
}

__global__ void cuZSKernel2(
	uchar* lpImage,
	uchar* lpSign,
	uchar* lpRef/*ˢ�±�־*/,
	ImType sw, ImType sh/*�߳�������*/)
{
	// ��^�滻�߼��ж�
	int bit[2] = { 8, 1 };

	int begin = 0, end = 0;

	ImType w = sw * gridDim.x + 2;

	ImType h = sh * blockDim.x + 2;

	int sum = 0, changed = 0;

	uchar sync[6] = { 0 };

	tagImPoint p[9] = { tagImPoint() }; // �����

	tagImPoint o(sw * blockIdx.x, sh * threadIdx.x, 0); // ��������ʼ��

	// ���򲹳�
	tagImPoint c[9] = {
		{ 0, 0, 0 }, { 0, -1, 0 }, { 0, -1, 1 },
		{ 0, 0, 1 }, { 0, 1, 1 }, { 0, 1, 0 },
		{ 0, 1, -1 }, { 0, 0, -1 }, { 0, -1, -1 } };

	for (ImType x = 0; x < sw; ++x)
	{
		for (ImType y = 0; y < sh; ++y)
		{
			p[0].x = o.x + x + 1;

			p[0].y = o.y + y + 1;// os

			p[0].idx = p[0].y * w + p[0].x;

			for (ImType b = 1; b <= 8; ++b)
			{
				p[0].b = b;

				for (int i = 1; i < 9; ++i)
				{
					p[i].x = p[0].x + c[i].x;

					p[i].y = p[0].y + c[i].y;

					p[i].b = p[0].b + c[i].b;

					// b = 0 -> b = 8, x = x -1
					// b = 9 -> b = 1, x = x + 1
					begin = !!!p[i].b;

					end = !(p[i].b ^ 9);

					p[i].x -= begin;

					p[i].x += end;

					p[i].b = (begin * bit[0]) ^ p[i].b;

					bit[0] = (begin * bit[0]) ^ p[i].b;

					p[i].b = (begin * bit[0]) ^ p[i].b;

					p[i].b = (end * bit[1]) ^ p[i].b;

					bit[1] = (end * bit[1]) ^ p[i].b;

					p[i].b = (end * bit[1]) ^ p[i].b;

					p[i].idx = p[i].y * w + p[i].x;

					sum += (lpImage[p[i].idx] >> (p[i].b - 1) & 0x1);

					bit[0] = 8; bit[1] = 1;
				} // 8����

				changed += max(0, (lpImage[p[1].idx] >> (p[1].b - 1) & 0x1) - (lpImage[p[8].idx] >> (p[8].b - 1) & 0x1));

				for (int i = 2; i < 9; ++i)
				{
					changed += max(0, (lpImage[p[i].idx] >> (p[i].b - 1) & 0x1) - (lpImage[p[i - 1].idx] >> (p[i - 1].b - 1) & 0x1));
				}

				sync[0] = !((lpImage[p[0].idx] >> (p[0].b - 1) & 0x1) ^ 1);// p1 = 1
				sync[1] = !(sum ^ min(max(2, sum), 6)); // 2 <= sum <=6
				sync[2] = !(changed ^ 1);// changed = 1
				sync[3] = !((lpImage[p[1].idx] >> (p[1].b - 1) & 0x1) *
					(lpImage[p[3].idx] >> (p[3].b - 1) & 0x1) *
					(lpImage[p[7].idx] >> (p[7].b - 1) & 0x1));// p2*p4*p8 = 0

				sync[4] = !((lpImage[p[7].idx] >> (p[7].b - 1) & 0x1) *
					(lpImage[p[1].idx] >> (p[1].b - 1) & 0x1) *
					(lpImage[p[5].idx] >> (p[5].b - 1) & 0x1));// p2*p6*p8 = 0

				sync[5] = (sync[0] & sync[1] & sync[2] & sync[3] & sync[4]);

				lpSign[p[0].idx] |= (sync[5] << (p[0].b - 1));

				lpRef[p[0].idx] |= (sync[5] << (p[0].b - 1));

				sum = 0, changed = 0;;
			}
		}
	}
}

// �����ڴ�48k
// 48 block 1024 thread
__global__ void cuZSSign(
	int* lpChanged,
	uchar* lpRef/*ˢ�±�־*/,
	ImType sw, ImType sh/*�߳�������*/)
{
	ImType w = sw * gridDim.x + 2;

	ImType h = sh * blockDim.x + 2;

	tagImPoint o(sw * blockIdx.x, sh * threadIdx.x, 0), p; // ��������ʼ��

	ImType idx = 0;

	for (ImType x = 0; x < sw; ++x)
	{
		for (ImType y = 0; y < sh; ++y)
		{
			idx = (o.y + y + 1) * w + o.x + x + 1;

			if (!!lpRef[idx])
			{
				atomicMax(lpChanged, (int)lpRef[idx]);
			}
		}
	}
}

// im = 73000*80000
// 700mb
// max int = 2gb(1,147,483,647) 
// max uint = 4gb(4,294,967,295)
// max long long = 8388607t(9,223,372,036,854,775,807)
// -> ���� = int
// 32g
void cuZhangSuen(Mat& cEncode, Mat& cImage, ImType nBlock, ImType nThread, ImType nSW, ImType nSH)
{
	ImType uImSize = (ImType)cEncode.rows * cEncode.cols;

	int nChanged[1] = { 0 }, *cuChanged = nullptr;

	uchar* cuImage = nullptr, * cuSign = nullptr, * cuRefresh = nullptr;

	hipMalloc(&cuSign, uImSize);

	hipMalloc(&cuImage, uImSize);

	hipMalloc(&cuRefresh, uImSize);

	hipMalloc(&cuChanged, sizeof(nChanged));

	hipMemcpy(cuImage, cEncode.data, uImSize, hipMemcpyHostToDevice);

	do
	{
		hipMemset(cuChanged, 0, sizeof(nChanged));

		hipMemset(cuSign, 0, uImSize);

		hipMemset(cuRefresh, 0, uImSize);

		cuZSKernel1 << <nBlock, nThread >> > (
			cuImage,
			cuSign,
			cuRefresh,
			nSW, nSH);
	
		cuZSSynchro << < nBlock, nThread >> > (
			cuImage, 
			cuSign,
			nSW, nSH);

		cuZSKernel2 << <nBlock, nThread >> > (
			cuImage,
			cuSign,
			cuRefresh,
			nSW, nSH);

		cuZSSynchro << < nBlock, nThread >> > (
			cuImage,
			cuSign,
			nSW, nSH);

		cuZSSign << <nBlock, nThread >> > (
			cuChanged,
			cuRefresh,
			nSW, nSH);

		hipMemcpy(nChanged, cuChanged, sizeof(nChanged), hipMemcpyDeviceToHost);

		cout << nChanged[0] << endl;

	} while (!!nChanged[0]);

	hipMemcpy(cEncode.data, cuImage, uImSize, hipMemcpyDeviceToHost);

	hipFree(cuChanged); hipFree(cuRefresh); hipFree(cuImage); hipFree(cuSign);
}

// ����ͼ����󣬸ò��ֲ�ȡ��x����ѹ��
int cuThinging(cv::Mat*& lpImage, cv::Mat*& lpThing)
{
	static ImType encode = 8;

	static ImType g_nBlock = 26 * 16; // 26 * 16

	static ImType g_nThread = 32 * 32; // 1024

	clock_t t1 = clock();

	cout << __FUNCTION__ << ": " << t1 << endl;

	ImType iImw = lpImage->cols, iImh = lpImage->rows;

	ImType w = ceil((double)iImw / encode / g_nBlock) * g_nBlock * encode;	// +1 �������

	ImType h = ceil((double)iImh / g_nThread) * g_nThread;

	// +1 �������
	// h - cImage.rows & w - cImage.cols �̶߳���	
	copyMakeBorder(*lpImage, *lpImage,
		1, h - iImh + 1, encode, w - iImw + encode, BORDER_CONSTANT, Scalar(0));

	Mat* lpEncode = new Mat(Mat::zeros(lpImage->rows, lpImage->cols * 0.125, CV_8UC1));

	ImEncode(*lpImage, *lpEncode);

	ImType nSW = lpEncode->cols / g_nBlock;

	ImType nSH = lpEncode->rows / g_nThread;

	cuZhangSuen(*lpEncode, *lpImage, g_nBlock, g_nThread, nSW, nSH);

	ImDecode(*lpEncode, *lpImage); delete lpEncode;

	(*lpThing) = (*lpImage)(Rect(encode, 1, iImw, iImh)).clone();

	clock_t t2 = clock();

	cout << __FUNCTION__ << ": " << (t2-t1) << endl;

	return 0;
}
